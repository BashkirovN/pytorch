#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/UnaryOps.h>

#include <limits>

#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/Math.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/NumericUtils.h>
#include <c10/core/Scalar.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <c10/util/complex.h>

namespace at {
namespace native {

void exp2_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "exp2_cuda",
      [&]() {
        gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::exp2(a);
        });
      });
}

// TODO: do we want this as a string or a resource string or ... ?
#define stringify(...) std::string(#__VA_ARGS__);
const auto i0_string = stringify(
  template <typename T>
  C10_HOST_DEVICE inline std::tuple<const T*, size_t> chebyshev_coefficients_i0e_A() {
    /* Chebyshev coefficients for exp(-x) I0(x)
    * in the interval [0,8].
    *
    * lim(x->0){ exp(-x) I0(x) } = 1.
    */
    static const T coefficients[] = {
        -4.41534164647933937950E-18, 3.33079451882223809783E-17,
        -2.43127984654795469359E-16, 1.71539128555513303061E-15,
        -1.16853328779934516808E-14, 7.67618549860493561688E-14,
        -4.85644678311192946090E-13, 2.95505266312963983461E-12,
        -1.72682629144155570723E-11, 9.67580903537323691224E-11,
        -5.18979560163526290666E-10, 2.65982372468238665035E-9,
        -1.30002500998624804212E-8,  6.04699502254191894932E-8,
        -2.67079385394061173391E-7,  1.11738753912010371815E-6,
        -4.41673835845875056359E-6,  1.64484480707288970893E-5,
        -5.75419501008210370398E-5,  1.88502885095841655729E-4,
        -5.76375574538582365885E-4,  1.63947561694133579842E-3,
        -4.32430999505057594430E-3,  1.05464603945949983183E-2,
        -2.37374148058994688156E-2,  4.93052842396707084878E-2,
        -9.49010970480476444210E-2,  1.71620901522208775349E-1,
        -3.04682672343198398683E-1,  6.76795274409476084995E-1};

    return std::make_tuple(coefficients, 30);
  }

  template <typename T>
  C10_HOST_DEVICE inline std::tuple<const T*, size_t> chebyshev_coefficients_i0e_B() {
    /* Chebyshev coefficients for exp(-x) sqrt(x) I0(x)
    * in the inverted interval [8,infinity].
    *
    * lim(x->inf){ exp(-x) sqrt(x) I0(x) } = 1/sqrt(2pi).
    */
    static const T coefficients[] = {
        -7.23318048787475395456E-18, -4.83050448594418207126E-18,
        4.46562142029675999901E-17,  3.46122286769746109310E-17,
        -2.82762398051658348494E-16, -3.42548561967721913462E-16,
        1.77256013305652638360E-15,  3.81168066935262242075E-15,
        -9.55484669882830764870E-15, -4.15056934728722208663E-14,
        1.54008621752140982691E-14,  3.85277838274214270114E-13,
        7.18012445138366623367E-13,  -1.79417853150680611778E-12,
        -1.32158118404477131188E-11, -3.14991652796324136454E-11,
        1.18891471078464383424E-11,  4.94060238822496958910E-10,
        3.39623202570838634515E-9,   2.26666899049817806459E-8,
        2.04891858946906374183E-7,   2.89137052083475648297E-6,
        6.88975834691682398426E-5,   3.36911647825569408990E-3,
        8.04490411014108831608E-1};

    return std::make_tuple(coefficients, 25);
  }

  template <typename scalar_t>
  static inline C10_HOST_DEVICE scalar_t
  chbevl(scalar_t _x, const scalar_t array[], size_t len) {
    using accscalar_t = at::acc_type<scalar_t, true>;

    accscalar_t x = static_cast<accscalar_t>(_x);
    accscalar_t b0, b1, b2;

    b0 = static_cast<accscalar_t>(array[0]);
    b1 = 0;

    for (size_t i = 1; i < len; ++i)  {
      b2 = b1;
      b1 = b0;
      b0 = x * b1 - b2 + static_cast<accscalar_t>(array[i]);
    }

    return static_cast<scalar_t>(0.5 * (b0 - b2));
  }

  template <typename scalar_t>
  static inline C10_HOST_DEVICE scalar_t calc_i0(scalar_t _x) {
    using accscalar_t = at::acc_type<scalar_t, true>;

    // Upcast input for numerical accuracy purposes
    // Needed for accurate results if input is bfloat16 or float16
    accscalar_t x = ::abs(static_cast<accscalar_t>(_x));

    if (x <= accscalar_t{8.0}) {
      auto coeff_pair = chebyshev_coefficients_i0e_A<accscalar_t>();
      auto A = std::get<0>(coeff_pair);
      auto len = std::get<1>(coeff_pair);
      accscalar_t y = (x / accscalar_t{2.0}) - accscalar_t{2.0};
      return static_cast<scalar_t>(::exp(x) * chbevl(y, A, len));
    }

    auto coeff_pair = chebyshev_coefficients_i0e_B<accscalar_t>();
    auto B = std::get<0>(coeff_pair);
    auto len = std::get<1>(coeff_pair);
    return static_cast<scalar_t>(::exp(x) * chbevl(accscalar_t{32.0} / x - accscalar_t{2.0}, B, len) / ::sqrt(x));
  }
); // stringify
#undef stringify

void i0_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "i0_cuda", [&]() {
    jitted_gpu_kernel</*return_dtype=*/ scalar_t, 
                      /*common_dtype=*/ scalar_t,
                      /*arity=*/ 1>(iter, i0_string);
    // gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
    //   return calc_i0(a);
    // });
  });
}

void i0e_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "i0e_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_i0e(a);
    });
  });
}

void i1_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.common_dtype(), "i1_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_i1(a);
    });
  });
}

void i1e_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.common_dtype(), "i1e_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_i1e(a);
    });
  });
}

void sigmoid_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "sigmoid_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return static_cast<scalar_t>(1) / (static_cast<scalar_t>(1) + std::exp(-a));
    });
  });
}

void sinc_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "sinc_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          if (a == scalar_t(0)) {
            return scalar_t(1);
          } else {
            // NVCC says constexpr var is not accessible from device
            scalar_t product = c10::detail::pi<scalar_t>() * a;
            return std::sin(product) / product;
          }
        });
      });
}

void logit_kernel_cuda(TensorIteratorBase& iter, const Scalar& eps_scalar) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.common_dtype(),
      "logit_cuda",
      [&]() {
        using T_ACC = acc_type<scalar_t, true>;
        const T_ACC eps = eps_scalar.to<T_ACC>();
        if (eps < T_ACC(0)) {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t x) -> scalar_t {
            const T_ACC x_acc = static_cast<T_ACC>(x);
            return c10::cuda::compat::log(x_acc / (T_ACC(1) - x_acc));
          });
        } else {
          const T_ACC lo = eps;
          const T_ACC hi = T_ACC(1) - eps;
          gpu_kernel(
              iter, [lo, hi] GPU_LAMBDA(scalar_t x) -> scalar_t {
                const T_ACC x_acc = static_cast<T_ACC>(x);
                T_ACC z = x_acc < lo ? lo : (x_acc > hi ? hi : x_acc);
                return c10::cuda::compat::log(z / (T_ACC(1) - z));
              });
        }
      });
}

void ndtri_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.common_dtype(), "ndtri_cuda", [&]() {
    gpu_kernel(
        iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t { return calc_ndtri(a); });
  });
}

void erf_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "erf_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::erf(a);
    });
  });
}

void erfc_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "erfc_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::erfc(a);
        });
      });
}

void erfinv_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "erfinv_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::erfinv(a);
    });
  });
}

void erfcx_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.common_dtype(), "erfcx_cuda", [&]() {
    gpu_kernel(
        iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t { return calc_erfcx(a); });
  });
}

void kaiser_window_kernel_cuda(TensorIteratorBase& iter, int64_t window_length, double beta_){
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.dtype(), "kaiser_window_cuda", [&](){
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC inv_alpha = static_cast<T_ACC>(2.0 / (window_length - 1));
    const T_ACC beta = static_cast<T_ACC>(beta_);
    const T_ACC inv_i0_beta = 1.0 / calc_i0(beta);
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t a) -> scalar_t {
      T_ACC x = static_cast<T_ACC>(a) * inv_alpha - 1;
      T_ACC y = std::max<T_ACC>(0, 1 - x * x);
      return calc_i0(beta * ::sqrt(y)) * inv_i0_beta;
    });
  });
}

void entr_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half,
      ScalarType::BFloat16,
      iter.common_dtype(),
      "entr_cuda",
      [&]() {
        gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t x) -> scalar_t {
          if (at::_isnan(x)) {
            return x;
          } else if (x > 0) {
            return -x * std::log(x);
          } else if (x == 0) {
            return 0;
          }
          return static_cast<scalar_t>(-INFINITY);
        });
      });
}

REGISTER_DISPATCH(exp2_stub, &exp2_kernel_cuda);
REGISTER_DISPATCH(i0_stub, &i0_kernel_cuda);
REGISTER_DISPATCH(special_i0e_stub, &i0e_kernel_cuda);
REGISTER_DISPATCH(special_i1_stub, &i1_kernel_cuda);
REGISTER_DISPATCH(special_i1e_stub, &i1e_kernel_cuda);
REGISTER_DISPATCH(sigmoid_stub, &sigmoid_kernel_cuda);
REGISTER_DISPATCH(sinc_stub, &sinc_kernel_cuda);
REGISTER_DISPATCH(logit_stub, &logit_kernel_cuda);
REGISTER_DISPATCH(erf_stub, &erf_kernel_cuda);
REGISTER_DISPATCH(erfc_stub, &erfc_kernel_cuda);
REGISTER_DISPATCH(erfinv_stub, &erfinv_kernel_cuda);
REGISTER_DISPATCH(kaiser_window_stub, &kaiser_window_kernel_cuda);
REGISTER_DISPATCH(special_entr_stub, &entr_kernel_cuda);
REGISTER_DISPATCH(special_ndtri_stub, &ndtri_kernel_cuda);
REGISTER_DISPATCH(special_erfcx_stub, &erfcx_kernel_cuda);

} // namespace native
} // namespace at
